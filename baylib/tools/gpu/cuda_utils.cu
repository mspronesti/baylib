#include "hip/hip_runtime.h"
//
// Created by paolo on 08/02/22.
//
#include "cuda_utils.cuh"

namespace baylib {

    template std::vector<uint> reduce_marginal_array(uint *arr, uint var_num, uint set_num);
    template std::vector<float> reduce_marginal_array(float *arr, uint var_num, uint set_num);

    /**
     * Kernel for accumulating multiple arrays into number of Blocks arrays
     * @param arr       : input array, will be overwritten by the result
     * @param var_num   : number of elements in a single array
     * @param n_set     : number of arrays
     */
     template <typename T>
    __global__ void reduce_marginal_array_kernel(T *arr, uint var_num, uint n_set) {
        extern __shared__ char shared_mem[];
        T* s_data = reinterpret_cast<T*>(shared_mem);
        uint id = threadIdx.x + blockDim.x * blockIdx.x;
        uint tid = threadIdx.x;

        // Load elements into shared memory
        for (uint i = 0; i < var_num; i++) {
            if (id < n_set)
                s_data[tid * var_num + i] = arr[id * var_num + i];
        }

        // Accumulate elemets into first array
        __syncthreads();
        for (uint s = 1; s < blockDim.x; s *= 2) {
            for (int i = 0; i < var_num; i++) {
                if (tid % (2 * s) == 0 && (tid + s) < blockDim.x && (id + s) < n_set) {
                    s_data[tid * var_num + i] += s_data[(tid + s) * var_num + i];
                }
            }
            __syncthreads();
        }

        // Copy array into the first elements of the output array
        if (tid == 0) {
            for (uint i = 0; i < var_num; i++) {
                arr[blockIdx.x * var_num + i] = s_data[i];
            }
        }

    }

    /**
     * Sample from a given discrete distribution while inside a cuda kernel
     * @tparam Probability  : Probability type of the distribution
     * @param distrib       : Distribution array
     * @param size          : size of the distribution
     * @param state         : CurandState for hiprand library
     * @return              : sample
     */
    template<typename Probability>
    __device__ uint discrete_sample(Probability *distrib, uint size, hiprandState *state) {
        auto sample = static_cast<Probability>(hiprand_uniform(state));
        uint i = 0;
        Probability prob = distrib[0];
        while (sample > prob && i < size)
            prob += distrib[++i];
        return i;
    }


    /**
     * Accumulate array of multiple marginal probability arrays into 1 single marginal vector
     * @param arr       : array of marginal array
     * @param var_num   : number of variables in the array
     * @param set_num   : number of arrays
     * @return          : marginal vector
     */
    template<typename T>
    std::vector<T> reduce_marginal_array(T *arr, uint var_num, uint set_num) {
        int max_shared_mem;
        size_t memory_for_single = var_num * sizeof(T);
        int device;
        int max_threads_per_block;
        ulong chucks;
        gpuErrcheck(hipGetDevice(&device));
        gpuErrcheck(hipDeviceGetAttribute(&max_shared_mem, hipDeviceAttributeMaxSharedMemoryPerBlock, device));
        gpuErrcheck(hipDeviceGetAttribute(&max_threads_per_block, hipDeviceAttributeMaxThreadsPerBlock, device));
        uint thread_capacity = max_shared_mem / memory_for_single;
        thread_capacity = thread_capacity > max_threads_per_block ? max_threads_per_block : thread_capacity;
        size_t shared_mem_size;
        chucks = set_num / thread_capacity + 1;

        shared_mem_size = var_num * thread_capacity * sizeof(T);
        reduce_marginal_array_kernel<<<chucks, thread_capacity, shared_mem_size >>>(arr, var_num, set_num);


        std::vector<T> marginal(var_num * set_num, 0);
        hipMemcpy(marginal.data(), arr, sizeof(T) * var_num * set_num, hipMemcpyDeviceToHost);

        if (chucks > 1) {
            for (int i = 1; i < chucks; i++) {
                for (int j = 0; j < var_num; j++) {
                    marginal[j] += marginal[i * var_num + j];
                }
            }
        }
        return marginal;
    }

    /**
     * Setup hiprandState for hiprand library
     * @param state : output array of dimension equal to the number of launched threads
     */
    __global__ void setup_kernel(hiprandState *state) {
        uint id = threadIdx.x + blockIdx.x * blockDim.x;
        /* Each thread gets different seed, a different sequence
           number, no offset */
        hiprand_init(7 + id, id, 0, &state[id]);
    }

    /**
     * Calculate kernel dimensions depending on the needed memory
     * @param samples               : Number of samples requested
     * @param shared_mem_per_thread : Memory needed by every single thread
     * @return                      : kernel parameters
     */
    baylib::kernel_params calc_kernel_parameters(uint samples, size_t shared_mem_per_thread) {
        int max_shared_mem;
        int device;
        baylib::kernel_params result{};
        int max_launchable_threads;
        int max_launchable_blocks;
        int warp_size;
        uint max_threads;
        uint n_threads;
        uint n_blocks;
        uint n_iter;

        gpuErrcheck(hipGetDevice(&device));
        gpuErrcheck(hipDeviceGetAttribute(&max_shared_mem, hipDeviceAttributeMaxSharedMemoryPerBlock, device));
        gpuErrcheck(hipDeviceGetAttribute(&max_launchable_threads, hipDeviceAttributeMaxThreadsPerBlock, device));
        gpuErrcheck(hipDeviceGetAttribute(&max_launchable_blocks, hipDeviceAttributeMaxGridDimX, device));
        gpuErrcheck(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device));

        max_threads = max_shared_mem / shared_mem_per_thread;
        n_threads = max_threads < max_launchable_threads ? max_threads : max_launchable_threads;
        if (n_threads > warp_size)
            n_threads -= n_threads % warp_size;
        result.N_Threads = n_threads;
        n_blocks = samples / n_threads + 1;
        n_blocks = n_blocks < max_launchable_blocks ? n_blocks : max_launchable_blocks;
        result.N_Blocks = n_blocks;
        n_iter = samples / (n_threads * n_blocks) + 1;
        result.N_Iter = n_iter;
        return result;
    }

}

